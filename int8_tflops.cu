#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define CHECK_CUDA(call)                                                          \
  do {                                                                             \
    hipError_t err = (call);                                                      \
    if (err != hipSuccess) {                                                      \
      fprintf(stderr, "CUDA error %s:%d: %s\n", __FILE__, __LINE__,                \
              hipGetErrorString(err));                                           \
      std::exit(1);                                                                \
    }                                                                              \
  } while (0)

#define CHECK_CUBLAS(call)                                                        \
  do {                                                                             \
    hipblasStatus_t st = (call);                                                    \
    if (st != HIPBLAS_STATUS_SUCCESS) {                                             \
      fprintf(stderr, "cuBLAS error %s:%d: %d\n", __FILE__, __LINE__, st);         \
      std::exit(2);                                                                \
    }                                                                              \
  } while (0)

int main(int argc, char** argv) {
  // parse args: N, timed_iters, warmup_iters
  int N           = (argc>1 ? std::atoi(argv[1]) : 65536);
  int timedIters  = (argc>2 ? std::atoi(argv[2]) : 80);
  int warmupIters = (argc>3 ? std::atoi(argv[3]) : 50);

  size_t bytesA = size_t(N)*N * sizeof(int8_t);
  size_t bytesC = size_t(N)*N * sizeof(int32_t);

  // allocate device matrices
  int8_t  *d_A, *d_B;
  int32_t *d_C;
  CHECK_CUDA(hipMalloc(&d_A, bytesA));
  CHECK_CUDA(hipMalloc(&d_B, bytesA));
  CHECK_CUDA(hipMalloc(&d_C, bytesC));

  // cuBLAS handle + Tensor-Core mode
  hipblasHandle_t handle;
  CHECK_CUBLAS(hipblasCreate(&handle));
  CHECK_CUBLAS(hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH));
  hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);


  const int32_t alpha = 1, beta = 0;

  // -- WARM-UP LOOPS --
  for (int i = 0; i < warmupIters; ++i) {
    CHECK_CUBLAS(hipblasGemmEx(handle,
      HIPBLAS_OP_N, HIPBLAS_OP_N,
      N, N, N,
      &alpha,
      d_A, HIP_R_8I, N,
      d_B, HIP_R_8I, N,
      &beta,
      d_C, HIP_R_32I, N,
      HIP_R_32I,
      CUBLAS_GEMM_DFALT_TENSOR_OP));
  }

  // setup timing
  hipEvent_t start, stop;
  CHECK_CUDA(hipEventCreate(&start));
  CHECK_CUDA(hipEventCreate(&stop));

  CHECK_CUDA(hipEventRecord(start, nullptr));
  for (int i = 0; i < timedIters; ++i) {
    CHECK_CUBLAS(hipblasGemmEx(handle,
      HIPBLAS_OP_N, HIPBLAS_OP_N,
      N, N, N,
      &alpha,
      d_A, HIP_R_8I, N,
      d_B, HIP_R_8I, N,
      &beta,
      d_C, HIP_R_32I, N,
      HIP_R_32I,
      CUBLAS_GEMM_DFALT_TENSOR_OP));
  }
  CHECK_CUDA(hipEventRecord(stop, nullptr));
  CHECK_CUDA(hipEventSynchronize(stop));

  float ms = 0;
  CHECK_CUDA(hipEventElapsedTime(&ms, start, stop));

  double secs   = ms/1000.0;
  double ops    = double(2ULL)*N*N*N*timedIters;
  double tops   = ops / secs / 1e12;

  printf("Matrix: %d×%d, Warm-ups: %d, Timed iters: %d\n",
         N, N, warmupIters, timedIters);
  printf("Elapsed: %.3f ms → %.2f TOPS (INT8)\n", ms, tops);

  // cleanup
  hipblasDestroy(handle);
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  return 0;
}
